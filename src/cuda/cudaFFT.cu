#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"


#define MAX_N 512

typedef struct {
    int size;
    double mat[MAX_N][MAX_N];
} Matrix;

typedef struct {
    int size;
    hipDoubleComplex mat[MAX_N][MAX_N];
} FreqMatrix;

void readMatrix(Matrix *m) {
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i][j]));
}

void printMatrix(FreqMatrix *m) {
    hipDoubleComplex sum = make_hipDoubleComplex(0, 0);
    for (int i = 0; i < m->size; i++) {
        for (int j = 0; j < m->size; j++) {
            sum = hipCadd(sum, m->mat[i][j]);
            printf("(%lf, %lf) ", hipCreal(m->mat[i][j]), hipCimag(m->mat[i][j]));
        }
        printf("\n");
    }
    sum = hipCdiv(sum, make_hipDoubleComplex(m->size, 0));
    printf("sum = (%lf, %lf)", hipCreal(sum), hipCimag(sum));
}

void transpose(FreqMatrix *m) {
    for (int i = 0; i < m->size; i++) {
        for (int j = i + 1; j < m->size; j++) {
            hipDoubleComplex tmp = m->mat[i][j];
            m->mat[i][j] = m->mat[j][i];
            m->mat[j][i] = tmp;
        }
    }
}

__global__ void fft_kernel(FreqMatrix *m, FreqMatrix *fm, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size) {
        fm->mat[i][j] = make_hipDoubleComplex(0, 0);
        for (int k = 0; k < size; k++) {
            double theta = -2 * M_PI * k * j / size;
            hipDoubleComplex w = make_hipDoubleComplex(cos(theta), sin(theta));
            fm->mat[i][j] = hipCadd(fm->mat[i][j], hipCmul(m->mat[i][k], w));
        }
        fm->mat[i][j] = hipCdiv(fm->mat[i][j], make_hipDoubleComplex(size, 0));
    }
}

__global__ void fft_kernel_transpose(FreqMatrix *m, FreqMatrix *fm, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size) {
        fm->mat[i][j] = make_hipDoubleComplex(0, 0);
        for (int k = 0; k < size; k++) {
            double theta = -2 * M_PI * k * i / size;
            hipDoubleComplex w = make_hipDoubleComplex(cos(theta), sin(theta));
            fm->mat[i][j] = hipCadd(fm->mat[i][j], hipCmul(m->mat[j][k], w));
        }
    }
    fm->mat[i][j] = hipCdiv(fm->mat[i][j], make_hipDoubleComplex(size, 0));
}

void fft(Matrix *mat, FreqMatrix *freq_domain) {
    int size = mat->size;
    freq_domain->size = size;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            freq_domain->mat[i][j] = make_hipDoubleComplex(mat->mat[i][j], 0);
        }
    }
    FreqMatrix *d_mat, *d_freq_domain;

    hipMalloc((void **)&d_mat, sizeof(FreqMatrix));
    hipMalloc((void **)&d_freq_domain, sizeof(FreqMatrix));
    hipMemcpy(d_mat, freq_domain, sizeof(FreqMatrix), hipMemcpyHostToDevice);
    hipMemcpy(d_freq_domain, freq_domain, sizeof(FreqMatrix), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                      (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    fft_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_mat, d_freq_domain, size);
    hipDeviceSynchronize();
    hipMemcpy(freq_domain, d_freq_domain, sizeof(FreqMatrix), hipMemcpyDeviceToHost);

    transpose(freq_domain);

    hipMemcpy(d_mat, freq_domain, sizeof(FreqMatrix), hipMemcpyHostToDevice);

    fft_kernel_transpose<<<blocksPerGrid, threadsPerBlock>>>(d_mat, d_freq_domain, size);
    hipDeviceSynchronize();
    hipMemcpy(freq_domain, d_freq_domain, sizeof(FreqMatrix), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_freq_domain);
}

int main() {
    Matrix mat;
    FreqMatrix freq_domain;
    readMatrix(&mat);
    fft(&mat, &freq_domain);
    printMatrix(&freq_domain);
    return 0;
}