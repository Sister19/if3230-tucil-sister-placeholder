// create dft program with cuda parallelization
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"


#define MAX_N 512

typedef struct {
    int size;
    double mat[MAX_N][MAX_N];
} Matrix;

typedef struct {
    int size;
    hipDoubleComplex mat[MAX_N][MAX_N];
} FreqMatrix;

void readMatrix(Matrix *m) {
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i][j]));
}

void printMatrix(FreqMatrix *m) {
    hipDoubleComplex sum = make_hipDoubleComplex(0, 0);
    for (int i = 0; i < m->size; i++) {
        for (int j = 0; j < m->size; j++) {
            sum = hipCadd(sum, m->mat[i][j]);
            printf("(%lf, %lf) ", hipCreal(m->mat[i][j]), hipCimag(m->mat[i][j]));
        }
        printf("\n");
    }
    sum = hipCdiv(sum, make_hipDoubleComplex(m->size, 0));
    printf("sum = (%lf, %lf)", hipCreal(sum), hipCimag(sum));
}

__device__ hipDoubleComplex handleElement(Matrix *mat, int k, int l, int i, int j) {
    double angle = 2 * M_PI * (i * k + j * l) / mat->size;
    hipDoubleComplex exp = make_hipDoubleComplex(cos(angle), -sin(angle));
    return hipCmul(make_hipDoubleComplex(mat->mat[i][j], 0), exp);
}

__device__ hipDoubleComplex handleRow(Matrix *mat, int k, int l, int i) {
    hipDoubleComplex sum = make_hipDoubleComplex(0, 0);
    for (int j = 0; j < mat->size; j++) {
        sum = hipCadd(sum, handleElement(mat, k, l, i, j));
    }
    return sum;
}

__device__ hipDoubleComplex handleColumn(Matrix *mat, int k, int l) {
    hipDoubleComplex sum = make_hipDoubleComplex(0, 0);
    for (int i = 0; i < mat->size; i++) {
        sum = hipCadd(sum, handleRow(mat, k, l, i));
    }
    return sum;
}

__device__ hipDoubleComplex dft(Matrix *mat, int k, int l) {
    hipDoubleComplex sum = handleColumn(mat, k, l);
    sum = hipCdiv(sum, make_hipDoubleComplex(mat->size * mat->size, 0));
    return sum;
}

__global__ void dft_kernel(Matrix *source, FreqMatrix *freq_domain) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int l = blockIdx.y * blockDim.y + threadIdx.y;
    if (k < source->size && l < source->size) {
        freq_domain->mat[k][l] = dft(source, k, l);
    }
}

void dft(Matrix *source, FreqMatrix *freq_domain) {
    freq_domain->size = source->size;
    Matrix *dev_source;
    FreqMatrix *dev_freq_domain;
    hipMalloc((void**)&dev_source, sizeof(Matrix));
    hipMalloc((void**)&dev_freq_domain, sizeof(FreqMatrix));
    hipMemcpy(dev_source, source, sizeof(Matrix), hipMemcpyHostToDevice);
    hipMemcpy(dev_freq_domain, freq_domain, sizeof(FreqMatrix), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((source->size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (source->size + threadsPerBlock.y - 1) / threadsPerBlock.y);
    dft_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_source, dev_freq_domain);
    hipDeviceSynchronize();
    hipMemcpy(freq_domain, dev_freq_domain, sizeof(FreqMatrix), hipMemcpyDeviceToHost);
    hipFree(dev_source);
    hipFree(dev_freq_domain);
}

int main() {
    Matrix source;
    FreqMatrix freq_domain;
    readMatrix(&source);
    dft(&source, &freq_domain);
    printMatrix(&freq_domain);
    return 0;
}



